

#ifndef __REDUCE_CU__
#define __REDUCE_CU__

#include "MarsInc.h"
#include "global.h"

__device__ void REDUCE_COUNT_FUNC//(void* key, void* vals, size_t keySize, size_t valCount)
{
	EMIT_COUNT_FUNC(sizeof(PVC_KEY_T), sizeof(PVC_VAL_T));
}

__device__ void REDUCE_FUNC//(void* key, void* vals, size_t keySize, size_t valCount)
{
	PVC_VAL_T* pVal = (PVC_VAL_T*)vals;
	pVal->phase = 1;
	EMIT_FUNC(key, vals, sizeof(PVC_KEY_T), sizeof(PVC_VAL_T));
}
#endif //__REDUCE_CU__
