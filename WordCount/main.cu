#include "hip/hip_runtime.h"


/******************************************************************
 *WordCount (WC): It counts the number of occurrences for each word in a file. Each Map
 * task processes a portion of the input file and emits intermediate data pairs, each of which consists
 * of a word as the key and a value of 1 for the occurrence. Group is required, and no reduce is
 * needed, because the Mars runtime provides the size of each group, after the Group stage.
 ******************************************************************/

#include "MarsInc.h"
#include "global.h"
#include <ctype.h>

#define __OUTPUT__

void validate(char* h_filebuf, Spec_t* spec, int num)
{
	char* key = (char*)spec->outputKeys;
	char* val = (char*)spec->outputVals;
	int4* offsetSizes = (int4*)spec->outputOffsetSizes;
	int2* range = (int2*)spec->outputKeyListRange;

	printf("# of words:%d\n", spec->outputDiffKeyCount);
	if (num > spec->outputDiffKeyCount) num = spec->outputDiffKeyCount;
	for (int i = 0; i < num; i++)
	{
		int keyOffset = offsetSizes[range[i].x].x;
		int valOffset = offsetSizes[range[i].x].z;
		char* word = key + keyOffset;
		int wordsize = *(int*)(val + valOffset);
		printf("%s - size: %d - count: %d\n", word, wordsize, range[i].y - range[i].x);
	}
}

//-----------------------------------------------------------------------
//usage: WordCount datafile
//param: datafile 
//-----------------------------------------------------------------------
int main( int argc, char** argv) 
{
	if (argc != 2)
	{
		printf("usage: %s datafile\n", argv[0]);
		exit(-1);	
	}
	
	Spec_t *spec = GetDefaultSpec();
	spec->workflow = MAP_GROUP;
#ifdef __OUTPUT__
	spec->outputToHost = 1;
#endif

	TimeVal_t allTimer;
	startTimer(&allTimer);

	TimeVal_t preTimer;
	startTimer(&preTimer);

	FILE* fp = fopen(argv[1], "r");
	fseek(fp, 0, SEEK_END);
	int fileSize = ftell(fp) + 1;
	rewind(fp);
	char* h_filebuf = (char*)malloc(fileSize);
	char* d_filebuf = NULL;
	fread(h_filebuf, fileSize, 1, fp);
	CUDA_SAFE_CALL(hipMalloc((void**)&d_filebuf, fileSize));	
	fclose(fp);

	WC_KEY_T key;
	key.file = d_filebuf;

	for (int i = 0; i < fileSize; i++)
		h_filebuf[i] = toupper(h_filebuf[i]);

	WC_VAL_T val;
	int offset = 0;
	char* p = h_filebuf;
	char* start = h_filebuf;
	while (1)
	{
		int blockSize = 2048;
		if (offset + blockSize > fileSize) blockSize = fileSize - offset;
		p += blockSize;
		for (; *p >= 'A' && *p <= 'Z'; p++);
			
		if (*p != '\0') 
		{
			*p = '\0'; 
			++p;
			blockSize = (int)(p - start);
			val.line_offset = offset;
			val.line_size = blockSize;
			AddMapInputRecord(spec, &key, &val, sizeof(WC_KEY_T), sizeof(WC_VAL_T));	
			offset += blockSize;
			start = p;
		}
		else
		{
			*p = '\0'; 
			blockSize = (int)(fileSize - offset);
			val.line_offset = offset;
			val.line_size = blockSize;
			AddMapInputRecord(spec, &key, &val, sizeof(WC_KEY_T), sizeof(WC_VAL_T));	
			break;
		}
	}
	CUDA_SAFE_CALL(hipMemcpy(d_filebuf, h_filebuf, fileSize, hipMemcpyHostToDevice));	
	endTimer("preprocess", &preTimer);
	//----------------------------------------------
	//map/reduce
	//----------------------------------------------
	MapReduce(spec);

	endTimer("all", &allTimer);
	//----------------------------------------------
	//further processing
	//----------------------------------------------
#ifdef __OUTPUT__
	CUDA_SAFE_CALL(hipMemcpy(h_filebuf, d_filebuf, fileSize, hipMemcpyDeviceToHost));	
	validate(h_filebuf, spec, 10);
#endif
	//----------------------------------------------
	//finish
	//----------------------------------------------
	FinishMapReduce(spec);
	hipFree(d_filebuf);
	free(h_filebuf);

	return 0;
}
