

#ifndef __MAP_CU__
#define __MAP_CU__

#include "MarsInc.h"
#include "global.h"

__device__ int hash_func(char* str, int len)
{
	int hash, i;
	for (i = 0, hash=len; i < len; i++)
		hash = (hash<<4)^(hash>>28)^str[i];
	return hash;
}

__device__ void MAP_COUNT_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{
	WC_KEY_T* pKey = (WC_KEY_T*)key;
	WC_VAL_T* pVal = (WC_VAL_T*)val;

	char* ptrBuf = pKey->file + pVal->line_offset;
	int line_size = pVal->line_size;

	char* p = ptrBuf;
	int lsize = 0;
	int wsize = 0;
	char* start = ptrBuf;

	while(1)
	{
		for (; *p >= 'A' && *p <= 'Z'; p++, lsize++);
		*p = '\0';
		++p;
		++lsize;
		wsize = (int)(p - start);
		if (wsize > 6)
		{
			//printf("%s, wsize:%d\n", start, wsize);	
			EMIT_INTER_COUNT_FUNC(wsize, sizeof(int));
		}
		for (; (lsize < line_size) && (*p < 'A' || *p > 'Z'); p++, lsize++);
		if (lsize >= line_size) break;
		start = p;
	}
}

__device__ void MAP_FUNC//(void *key, void val, size_t keySize, size_t valSize)
{
	WC_KEY_T* pKey = (WC_KEY_T*)key;
	WC_VAL_T* pVal = (WC_VAL_T*)val;

	char* filebuf = pKey->file;
	char* ptrBuf = filebuf + pVal->line_offset;
	int line_size = pVal->line_size;

	char* p = ptrBuf;
	char* start = ptrBuf;
	int lsize = 0;
	int wsize = 0;

	while(1)
	{
		for (; *p >= 'A' && *p <= 'Z'; p++, lsize++);
		*p = '\0';
		++p;
		++lsize;
		wsize = (int)(p - start);
		int* o_val = (int*)GET_OUTPUT_BUF(0);
		*o_val = wsize;
		if (wsize > 6) 
		{
			//printf("%s, %d\n", start, wsize);	
			EMIT_INTERMEDIATE_FUNC(start, o_val, wsize, sizeof(int));
		}
		for (; (lsize < line_size) && (*p < 'A' || *p > 'Z'); p++, lsize++);
		if (lsize >= line_size) break;
		start = p;	
	}
}
#endif //__MAP_CU__
