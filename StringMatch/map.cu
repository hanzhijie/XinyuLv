#ifndef __MAP_CU__
#define __MAP_CU__

#include "MarsInc.h"
#include "global.h"

__device__ void MAP_COUNT_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{
	SM_KEY_T* pKey = (SM_KEY_T*)key;
	SM_VAL_T* pVal = (SM_VAL_T*)val;

	int bufOffset = pVal->linebuf_offset;
	int bufSize = pVal->linebuf_size;
	char* buf = pKey->ptrFile + bufOffset;

	char* keyword =  pKey->ptrKeyword;
	int keywordSize = pVal->keyword_size;

	int cur = 0;
	char* p = buf;
	char* start = buf;

	while(1)
	{
		for (; *p != '\n'; ++p, ++cur);
		++p;
		int wordSize = (int)(p - start);

		if (cur >= bufSize) break;
		char* k = keyword;
		char* s = start;
		if (wordSize == keywordSize) 
		{
			for (; *s == *k && *k != '\0'; s++, k++);
			if (*s == '\n') EMIT_INTER_COUNT_FUNC(sizeof(int), sizeof(int));
		}

		start = p;
		bufOffset += wordSize;
	}
}

__device__ void MAP_FUNC//(void *key, void val, size_t keySize, size_t valSize)
{
	SM_KEY_T* pKey = (SM_KEY_T*)key;
	SM_VAL_T* pVal = (SM_VAL_T*)val;

	int bufOffset = pVal->linebuf_offset;
	int bufSize = pVal->linebuf_size;
	char* buf = pKey->ptrFile + bufOffset;

	char* keyword =  pKey->ptrKeyword;
	int keywordSize = pVal->keyword_size;

	int cur = 0;
	char* p = buf;
	char* start = buf;

	while(1)
	{
		for (; *p != '\n'; ++p, ++cur);
		++p;
		int wordSize = (int)(p - start);
		int wordOffset = bufOffset;

		if (cur >= bufSize) break;

		char* k = keyword;
		char* s = start;

		if (wordSize == keywordSize) 
		{
			for (; *s == *k && *k != '\0'; s++, k++);
			if (*s == '\n') 
			{
				int* o_offset = (int*)GET_OUTPUT_BUF(0);
				int* o_size = (int*)GET_OUTPUT_BUF(sizeof(int));
				*o_offset = wordOffset;
				*o_size = wordSize;
				EMIT_INTERMEDIATE_FUNC(o_offset, o_size, sizeof(int), sizeof(int));
			}
		}

		start = p;
		bufOffset += wordSize;
	}
}
#endif //__MAP_CU__
