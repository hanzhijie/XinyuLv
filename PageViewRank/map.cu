
#ifndef __MAP_CU__
#define __MAP_CU__

#include "MarsInc.h"

__device__ void MAP_COUNT_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{
	EMIT_INTER_COUNT_FUNC(sizeof(int), sizeof(int));
}

__device__ void MAP_FUNC//(void *key, void val, size_t keySize, size_t valSize)
{
	int* o_rank = (int*)GET_OUTPUT_BUF(0);
	int* o_offset = (int*)GET_OUTPUT_BUF(sizeof(int));
	*o_rank = *(int*)key;
	*o_offset = *(int*)val;
	EMIT_INTERMEDIATE_FUNC(o_rank, o_offset, sizeof(int), sizeof(int));
}
#endif //__MAP_CU__
