
/********************************************************************
 *Page View Rank (PVR): With the output of the
 *Page View Count, the Map in Page View Rank takes
 *the pair of the page access count as the key and the
 *URL as the value, and obtains the top ten URLs that are
 *most frequently accessed. No Reduce stage is required.
 ********************************************************************/

#include "MarsInc.h"

#define __OUTPUT__

void printFun(void* key, void* val, int keySize, int valSize)
{
	int count = *(int*)key;
	int offset = *(int*)val;

	printf("count: %d, offset: %d\n", count, offset);
}

void validate(Spec_t* spec, int num)
{
	PrintOutputRecords(spec, num, printFun);
}

//-----------------------------------------------------------------
//usage: PageViewRank datafile 
//param: datafile
//-----------------------------------------------------------------
int main( int argc, char** argv) 
{
	if (argc != 2) 
	{
		printf("usage: %s filename\n", argv[0]);
		exit(-1);
	}

	TimeVal_t timer;
	startTimer(&timer);

	Spec_t *spec = GetDefaultSpec();
	spec->workflow = MAP_GROUP;
#ifdef __OUTPUT__
	spec->outputToHost = 1;
#endif
		
	//-----------------------------------------------------
	//make map input
	//-----------------------------------------------------
	TimeVal_t loadtimer;
	startTimer(&loadtimer);
	char *filename = argv[1];
	FILE* fp = fopen(filename, "r");
	fseek(fp, 0, SEEK_END);
	int fileSize = ftell(fp) + 1;
	rewind(fp);
    	char *h_filebuf = (char*)malloc(fileSize);
	fread(h_filebuf, fileSize, 1, fp);
	fclose(fp);

	int offset = 0;
	char* p = h_filebuf;
	char* start = h_filebuf;
	int cur = 0;
	while (1)
	{
		int lineSize = 0;
		for (; *p != '\t'; ++p, ++lineSize, ++cur);
		char* rankString = p + 1;
		for (; *p != '\n' && *p != '\0'; ++p, ++lineSize, ++cur);	

		*p = '\0';
		++p;
		lineSize = (int)(p - start);
		int rank = atoi(rankString);
		//printf("%s\n", rankString);
		AddMapInputRecord(spec, &rank, &offset, sizeof(int), sizeof(int));	
		offset += lineSize; 
		start = p;

		if (offset >= fileSize-1) break;
	}
	endTimer("io-test", &loadtimer);

	//------------------------------------------------------
	//main MapReduce procedure
	//------------------------------------------------------
	MapReduce(spec);

	//------------------------------------------------------
	//further processing
	//------------------------------------------------------
#ifdef __OUTPUT__
	validate(spec, 10);
#endif

	//------------------------------------------------------
	//finish
	//------------------------------------------------------
	FinishMapReduce(spec);
	free(h_filebuf);
	endTimer("all-test", &timer);
	return 0;
}
