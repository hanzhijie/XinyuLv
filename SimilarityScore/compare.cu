#include "hip/hip_runtime.h"

#ifndef __COMPARE_CU__
#define __COMPARE_CU__
#include "MarsInc.h"

#define SMALL	0.0000001f
__device__ int compare(const void *d_a, int len_a, const void *d_b, int len_b)
{
	float a = *(float*)d_a;
	float b = *(float*)d_b;

	if (fabsf(a-b) < SMALL)  return 0;
	if (a < b) return 1;

	return -1;
}

#endif //__COMPARE_CU__
