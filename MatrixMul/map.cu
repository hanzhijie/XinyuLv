#include "hip/hip_runtime.h"

#ifndef __MAP_CU__
#define __MAP_CU__

#include "MarsInc.h"
#include "global.h"

__device__ float operator*(float4 a, float4 b)
{
	return (a.x*b.x+a.y*b.y+a.z*b.z+a.w*b.w);
}

__device__ void MAP_COUNT_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{			
	EMIT_INTER_COUNT_FUNC(sizeof(float), sizeof(int2));
}

__device__ void MAP_FUNC//(void *key, void val, size_t keySize, size_t valSize)
{
	MM_KEY_T* pKey = ((MM_KEY_T*)key);
	MM_VAL_T* pVal = ((MM_VAL_T*)val);

	int rowId = pVal->row;
	int colId = pVal->col;

	int M_COL_COUNT = pVal->col_dim;

	float4 *matrix1 = (float4*)(pKey->matrix1+rowId*M_COL_COUNT);
	float4 *matrix2 = (float4*)(pKey->matrix2+colId*M_COL_COUNT);

	float newVal = 0.0f;

	int col4 = M_COL_COUNT >> 2;
	int remainder = M_COL_COUNT & 0x00000003;

	for (int i = 0; i < col4; i++)
	{
		float4 v1 = matrix1[i];
		float4 v2 = matrix2[i];

		newVal += v1.x * v2.x;
		newVal += v1.y * v2.y;
		newVal += v1.z * v2.z;
		newVal += v1.w * v2.w;
	}

	float *rMatrix1 = (float*)(matrix1+col4);
	float *rMatrix2 = (float*)(matrix2+col4);

	for (int i = 0; i < remainder; i++)
	{
		float f1 = rMatrix1[i];
		float f2 = rMatrix2[i];
		newVal += (f1 * f2);
	}

	float* o_result = (float*)GET_OUTPUT_BUF(0);
	*o_result = newVal;
	int2* o_pos = (int2*)GET_OUTPUT_BUF(sizeof(float));
	o_pos->x = rowId;
	o_pos->y = colId;
	EMIT_INTERMEDIATE_FUNC(o_result, o_pos, sizeof(float), sizeof(int2));			 
}

#endif //__MAP_CU__
