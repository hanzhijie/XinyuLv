

#ifndef __REDUCE_CU__
#define __REDUCE_CU__

#include "MarsInc.h"
#include "global.h"

//-------------------------------------------------------------------------
//No Reduce in this application
//-------------------------------------------------------------------------
__device__ void REDUCE_COUNT_FUNC//(void* key, void* vals, size_t keySize, size_t valCount)
{
	EMIT_COUNT_FUNC(sizeof(KM_KEY_T), sizeof(KM_VAL_T));
}

__device__ void REDUCE_FUNC//(void* key, void* vals, size_t keySize, size_t valCount)
{
	KM_KEY_T* pKey = (KM_KEY_T*)key;
	KM_VAL_T* pFirstVal = (KM_VAL_T*)vals;
	int dim = pKey->dim;
	int firstPtId = pKey->point_id;
	int cluster_id = pKey->ptrClusterId[firstPtId];
	int* clusters = (int*)pFirstVal->ptrClusters + cluster_id * dim;
	int* points = (int*)pFirstVal->ptrPoints;

	for (int i = 0; i < dim; i++)
		clusters[i] = 0;

	for (int i = 0; i < valCount; i++)
	{
		KM_KEY_T* iKey = (KM_KEY_T*)GET_KEY_FUNC(key, i);
		int* pt = points + iKey->point_id * dim;	
		for (int j = 0; j < dim; j++)
			clusters[j] += pt[j];	
	}

	for (int i = 0; i < dim; i++)
		clusters[i] /= (int)valCount;

	//EMIT_FUNC(key, vals, sizeof(KM_KEY_T), sizeof(KM_VAL_T));
}
#endif //__REDUCE_CU__
